#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <direct.h>
#include <helper_gl.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>          // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <vec.h>
/***********************delete********************/
#include <fstream>
#include <iostream>
/***********************delete********************/
#define _MAX_FLOAT_CNT_PER_BLOCK_SHARED_MEM_ 12288
#define _MAX_LIGHT_CNT_PER_PROCESS_ 512   //32 * 16
#define _UINT_CNT_PER_PIXEL_RESULT_ 32        //��ʱֻ֧��1024����

/**************************DEVICE FUNCTION*************************/
__inline__ __device__ float dot3(const float3& a, const float3& b){
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__inline__ __device__ float3 operator- (const float3 &a, const float3 &b){
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__inline__ __device__ float3 operator+ (const float3 &a, const float3 &b){
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__inline__ __device__ float3 operator* (const float3 &v, float b){
	return make_float3(v.x*b, v.y*b, v.z*b);
}

__inline__ __device__ float3 normalize(const float3& v){
	float len_inv = 1.0f / sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
	return make_float3(v.x*len_inv, v.y*len_inv, v.z*len_inv);
}
template <typename T>
__inline__ __device__ T mclamp(T a, T b, T c){
	if (a < b) return b;  if (a>c) return c; return a;
};

__device__ unsigned int cnt1(unsigned int number){
	unsigned int ret = 0;
	while (number > 0){
		if (number & 1u) ret++;
		number >>= 1u;
	}
	return ret;
};
/**************************GLOBAL FUNCTION*************************/
texture<float4, hipTextureType2D, hipReadModeElementType> pos_ref;
texture<float4, hipTextureType2D, hipReadModeElementType> norm_ref;

__global__ void calVisibility_compressed(int width, int height, int lightCnt, int slight, float3* glights, int rot_xy, float3* gaxises, float3* gradius, unsigned int *columns, unsigned int* indexes, float3 center, unsigned int precision, unsigned int *result){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int tid_inblock_const = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned int threads_num_per_block = blockDim.x*blockDim.y;
	const unsigned int res2 = 1u << (precision - 1);
	const unsigned int resolution = 1u << precision;

	__shared__ float3 lights_sm[_MAX_LIGHT_CNT_PER_PROCESS_];
	{
		unsigned int tid_inblock = tid_inblock_const;
		for (; tid_inblock < lightCnt; tid_inblock += threads_num_per_block){
			lights_sm[tid_inblock] = glights[tid_inblock] - center;
		}
	}
	__syncthreads();

	if (x < width&&y < height){
		float4 tmpf4 = tex2D(norm_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 pnorm = make_float3(tmpf4.x, tmpf4.y, tmpf4.z);
#define _poffset_ 0.6f
		tmpf4 = tex2D(pos_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 ppos = make_float3(tmpf4.x + pnorm.x*_poffset_ - center.x, tmpf4.y + pnorm.y*_poffset_ - center.y, tmpf4.z + pnorm.z*_poffset_ - center.z);

		unsigned int xdir, ydir, rotID;
		float3 axis_x, axis_y, axis_z, radius;
		unsigned int *resultptr = result + (y*width + x)*_UINT_CNT_PER_PIXEL_RESULT_;
		for (int l = 0; l < lightCnt; l++){
			float3 lpos = lights_sm[l];
			float3 ray = lpos - ppos;

			if (dot3(ray, pnorm)>0.0){
				{//find most matching coordinate-system
					ray = normalize(ray);
					float cosx = sqrt(ray.x*ray.x + ray.z*ray.z);
					if (cosx < 1e-6){ xdir = rot_xy / 2, ydir = 0; }
					else{
						if (ray.x>0) cosx = -cosx;
						xdir = unsigned int(acos(cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						ydir = unsigned int(acos(ray.z / cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						xdir = mclamp<unsigned int>(xdir, 0, rot_xy - 1);
						ydir = mclamp<unsigned int>(ydir, 0, rot_xy - 1);
					}
					rotID = (ydir*rot_xy + xdir);
					radius = gradius[rotID];
					axis_x = gaxises[rotID * 3 + 0];
					axis_y = gaxises[rotID * 3 + 1];
					axis_z = gaxises[rotID * 3 + 2];
				}
				float3 midp = (ppos + ray + lpos) * 0.5f;      ////////////////////////////////////////////!!!!!!!!!!!!!!!!!!!!!!!!!! + ray
				unsigned int midx = unsigned int((dot3(midp, axis_x) / radius.x + 1)*res2);
				unsigned int midy = unsigned int((dot3(midp, axis_y) / radius.y + 1)*res2);

				if (midx < 0 || midx >= resolution || midy < 0 || midy >= resolution){ // line segment didnt intersect with scene.
					resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
				}
				else{
					unsigned int z1 = unsigned int((dot3(ppos + ray, axis_z) / radius.z + 1)*res2); //////////////////////////////!!!!!!!!!!!!!!!!!!!+ray
					unsigned int z2 = unsigned int((dot3(lpos, axis_z) / radius.z + 1)*res2);
					if (z1>z2) { unsigned int tmp = z1; z1 = z2; z2 = tmp; }

					if (z1 < resolution && z2 >= 0){
						if (z1 < 0) z1 = 0;
						if (z2 >= resolution) z2 = resolution - 1;
						//unsigned int * tptr = &(gtags[(rotID << (precision * 3 - 5)) + (midx << (precision * 2 - 5)) + (midy << (precision - 5))]);
						unsigned int * tptr = &columns[indexes[((rotID << (precision << 1u)) + (midx << precision) + midy)] << (precision - 5)];

						unsigned int zs = z1 >> 5;
						unsigned int ze = z2 >> 5;
						unsigned int m1 = z1 & 31u;
						unsigned int m2 = z2 & 31u;
						if (zs == ze){
							//if (!((tptr[zs] << (31 - m2)) >> (31 - m2 + m1))) resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
							if (!((tptr[zs] << (32 - m2)) >> (32 - m2 + m1))) resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
						}
						else{
							bool flag = false;
							for (int c = zs + 1; c < ze; c++) if (tptr[c]){ flag = true; break; };
							if (!flag){
								if ((!((~((1u << (m1 + 1)) - 1u))&(tptr[zs]))) && (!(((1u << (m2)) - 1u)&(tptr[ze])))){
									resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
								}
							}
						}
					}
					else {
						resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
					}
				}
			}
		}
	}
}

__global__ void genShadowMap_compressed(int width, int height, int lightCnt, int slight, float3* glights, int rot_xy, float3* gaxises, float3* gradius, unsigned int *columns, unsigned int* indexes, float3 center, unsigned int precision, unsigned int* ssbo_counter){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int tid_inblock_const = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned int threads_num_per_block = blockDim.x*blockDim.y;
	const unsigned int res2 = 1u << (precision - 1);
	const unsigned int resolution = 1u << precision;

	__shared__ float3 lights_sm[_MAX_LIGHT_CNT_PER_PROCESS_];
	{
		unsigned int tid_inblock = tid_inblock_const;
		for (; tid_inblock < lightCnt; tid_inblock += threads_num_per_block){
			lights_sm[tid_inblock] = glights[tid_inblock] - center;
		}
	}
	__syncthreads();

	if (x < width&&y < height){
		float4 tmpf4 = tex2D(norm_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 pnorm = make_float3(tmpf4.x, tmpf4.y, tmpf4.z);
#define _poffset_ 0.6f
		tmpf4 = tex2D(pos_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 ppos = make_float3(tmpf4.x + pnorm.x*_poffset_ - center.x, tmpf4.y + pnorm.y*_poffset_ - center.y, tmpf4.z + pnorm.z*_poffset_ - center.z);

		unsigned int xdir, ydir, rotID;
		float3 axis_x, axis_y, axis_z, radius;

		unsigned int littedCnt = 0;
		for (int l = 0; l < lightCnt; l++){
			float3 lpos = lights_sm[l];
			float3 ray = lpos - ppos;

			if (dot3(ray, pnorm)>0.0){
				{   //find most matching coordinate-system
					ray = normalize(ray);
					float cosx = sqrt(ray.x*ray.x + ray.z*ray.z);
					if (cosx < 1e-6){ xdir = rot_xy / 2, ydir = 0; }
					else{
						if (ray.x>0) cosx = -cosx;
						xdir = unsigned int(acos(cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						ydir = unsigned int(acos(ray.z / cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						xdir = mclamp<unsigned int>(xdir, 0, rot_xy - 1);
						ydir = mclamp<unsigned int>(ydir, 0, rot_xy - 1);
					}
					rotID = (ydir*rot_xy + xdir);
					radius = gradius[rotID];
					axis_x = gaxises[rotID * 3 + 0];
					axis_y = gaxises[rotID * 3 + 1];
					axis_z = gaxises[rotID * 3 + 2];
				}
				float3 midp = (ppos + ray + lpos) * 0.5f;      ////////////////////////////////////////////!!!!!!!!!!!!!!!!!!!!!!!!!! + ray
				unsigned int midx = unsigned int((dot3(midp, axis_x) / radius.x + 1)*res2);
				unsigned int midy = unsigned int((dot3(midp, axis_y) / radius.y + 1)*res2);

				if (midx < 0 || midx >= resolution || midy < 0 || midy >= resolution){ // line segment didnt intersect with scene.
					littedCnt++;
				}
				else{
					unsigned int z1 = unsigned int((dot3(ppos + ray, axis_z) / radius.z + 1)*res2); //////////////////////////////!!!!!!!!!!!!!!!!!!!+ray
					unsigned int z2 = unsigned int((dot3(lpos, axis_z) / radius.z + 1)*res2);
					if (z1>z2) { unsigned int tmp = z1; z1 = z2; z2 = tmp; }

					if (z1 < resolution && z2 >= 0){
						if (z1 < 0) z1 = 0;
						if (z2 >= resolution) z2 = resolution - 1;
						unsigned int * tptr = &columns[indexes[((rotID << (precision << 1u)) + (midx << precision) + midy)] << (precision - 5)];

						unsigned int zs = z1 >> 5;  unsigned int ze = z2 >> 5;
						unsigned int m1 = z1 & 31u;  unsigned int m2 = z2 & 31u;
						if (zs == ze){ if (!((tptr[zs] << (32 - m2)) >> (32 - m2 + m1))) littedCnt++; }
						else{
							bool flag = false;
							for (int c = zs + 1; c < ze; c++) if (tptr[c]){ flag = true; break; };
							if (!flag){
								if ((!((~((1u << (m1 + 1)) - 1u))&(tptr[zs]))) && (!(((1u << (m2)) - 1u)&(tptr[ze])))){ littedCnt++; }
							}
						}
					}
					else littedCnt++;
				}
			}
		}
		ssbo_counter[y*width + x] += littedCnt;
	}
}

__global__ void calVisibility_uncompressed(int width, int height, int lightCnt, int slight, float3* glights, int rot_xy, float3* gaxises, float3* gradius, unsigned int *gtags, float3 center, unsigned int precision, unsigned int *result){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int tid_inblock_const = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned int threads_num_per_block = blockDim.x*blockDim.y;
	const unsigned int res2 = 1u << (precision - 1);
	const unsigned int resolution = 1u << precision;

	__shared__ float3 lights_sm[_MAX_LIGHT_CNT_PER_PROCESS_];
	{
		unsigned int tid_inblock = tid_inblock_const;
		for (; tid_inblock < lightCnt; tid_inblock += threads_num_per_block){
			lights_sm[tid_inblock] = glights[tid_inblock] - center;
		}
	}
	__syncthreads();

	if (x < width&&y < height){
		float4 tmpf4 = tex2D(norm_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 pnorm = make_float3(tmpf4.x, tmpf4.y, tmpf4.z);
#define _poffset_ 0.6f
		tmpf4 = tex2D(pos_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 ppos = make_float3(tmpf4.x + pnorm.x*_poffset_ - center.x, tmpf4.y + pnorm.y*_poffset_ - center.y, tmpf4.z + pnorm.z*_poffset_ - center.z);

		unsigned int xdir, ydir, rotID;
		float3 axis_x, axis_y, axis_z, radius;
		unsigned int *resultptr = result + (y*width + x)*_UINT_CNT_PER_PIXEL_RESULT_;
		for (int l = 0; l < lightCnt; l++){
			float3 lpos = lights_sm[l];
			float3 ray = lpos - ppos;

			if (dot3(ray, pnorm)>0.0){
				{   //find most matching coordinate-system
					ray = normalize(ray);
					float cosx = sqrt(ray.x*ray.x + ray.z*ray.z);
					if (cosx < 1e-6){ xdir = rot_xy / 2, ydir = 0; }
					else{
						if (ray.x>0) cosx = -cosx;
						xdir = unsigned int(acos(cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						ydir = unsigned int(acos(ray.z / cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						xdir = mclamp<unsigned int>(xdir, 0, rot_xy - 1);
						ydir = mclamp<unsigned int>(ydir, 0, rot_xy - 1);
					}
					rotID = (ydir*rot_xy + xdir);
					radius = gradius[rotID];
					axis_x = gaxises[rotID * 3 + 0];
					axis_y = gaxises[rotID * 3 + 1];
					axis_z = gaxises[rotID * 3 + 2];
				}
				float3 midp = (ppos + ray + lpos) * 0.5f;      ////////////////////////////////////////////!!!!!!!!!!!!!!!!!!!!!!!!!! + ray
				unsigned int midx = unsigned int((dot3(midp, axis_x) / radius.x + 1)*res2);
				unsigned int midy = unsigned int((dot3(midp, axis_y) / radius.y + 1)*res2);

				if (midx < 0 || midx >= resolution || midy < 0 || midy >= resolution){ // line segment didnt intersect with scene.
					resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
				}
				else{
					unsigned int z1 = unsigned int((dot3(ppos + ray, axis_z) / radius.z + 1)*res2); //////////////////////////////!!!!!!!!!!!!!!!!!!!+ray
					unsigned int z2 = unsigned int((dot3(lpos, axis_z) / radius.z + 1)*res2);
					if (z1>z2) { unsigned int tmp = z1; z1 = z2; z2 = tmp; }

					if (z1 < resolution && z2 >= 0){   // line segment didnt intersect with scene.
						if (z1 < 0) z1 = 0;
						if (z2 >= resolution) z2 = resolution - 1;
						unsigned int * tptr = &(gtags[(rotID << (precision * 3 - 5)) + (midx << (precision * 2 - 5)) + (midy << (precision - 5))]);

						unsigned int zs = z1 >> 5;  unsigned int ze = z2 >> 5;
						unsigned int m1 = z1 & 31u;  unsigned int m2 = z2 & 31u;
						if (zs == ze){
							//if (!((tptr[zs] << (31 - m2)) >> (31 - m2 + m1))) resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
							if (!((tptr[zs] << (32 - m2)) >> (32 - m2 + m1))) resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
						}
						else{
							bool flag = false;
							for (int c = zs + 1; c < ze; c++) if (tptr[c]){ flag = true; break; };
							if (!flag){
								if ((!((~((1u << (m1 + 1)) - 1u))&(tptr[zs]))) && (!(((1u << (m2)) - 1u)&(tptr[ze])))){
									//if (x == 167 && y == 275 && l == 10)printf("im here");
									//if ((!((~((1u << m1) - 1u))&(tptr[zs]))) && (!(((1u << (m2 + 1)) - 1u)&(tptr[ze])))){
									///////////////////////////////////////////////////////////////////////////write result////////////////////////////////////////////////////////////////
									resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
								}
							}
						}
					}
					else{
						resultptr[(l + slight) >> 5u] |= (1u << ((l + slight) & 31u));
					}
				}
			}
		}
	}
}

__global__ void genShadowMap_uncompressed(int width, int height, int lightCnt, int slight, float3* glights, int rot_xy, float3* gaxises, float3* gradius, unsigned int *gtags, float3 center, unsigned int precision, unsigned int* ssbo_counter){
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int tid_inblock_const = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned int threads_num_per_block = blockDim.x*blockDim.y;
	const unsigned int res2 = 1u << (precision - 1);
	const unsigned int resolution = 1u << precision;

	__shared__ float3 lights_sm[_MAX_LIGHT_CNT_PER_PROCESS_];
	{
		unsigned int tid_inblock = tid_inblock_const;
		for (; tid_inblock < lightCnt; tid_inblock += threads_num_per_block){
			lights_sm[tid_inblock] = glights[tid_inblock] - center;
		}
	}
	__syncthreads();

	if (x < width&&y < height){
		float4 tmpf4 = tex2D(norm_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 pnorm = make_float3(tmpf4.x, tmpf4.y, tmpf4.z);
#define _poffset_ 0.6f
		tmpf4 = tex2D(pos_ref, (x + 0.5f) / (float(width)), (y + 0.5f) / (float(height)));
		const float3 ppos = make_float3(tmpf4.x + pnorm.x*_poffset_ - center.x, tmpf4.y + pnorm.y*_poffset_ - center.y, tmpf4.z + pnorm.z*_poffset_ - center.z);

		unsigned int xdir, ydir, rotID;
		float3 axis_x, axis_y, axis_z, radius;
		unsigned int littedCnt = 0;
		for (int l = 0; l < lightCnt; l++){
			float3 lpos = lights_sm[l];
			float3 ray = lpos - ppos;

			if (dot3(ray, pnorm)>0.0){
				{   //find most matching coordinate-system
					ray = normalize(ray);
					float cosx = sqrt(ray.x*ray.x + ray.z*ray.z);
					if (cosx < 1e-6){ xdir = rot_xy / 2, ydir = 0; }
					else{
						if (ray.x>0) cosx = -cosx;
						xdir = unsigned int(acos(cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						ydir = unsigned int(acos(ray.z / cosx) * 57.29578f / 180.0f * rot_xy + 0.5f);
						xdir = mclamp<unsigned int>(xdir, 0, rot_xy - 1);
						ydir = mclamp<unsigned int>(ydir, 0, rot_xy - 1);
					}
					rotID = (ydir*rot_xy + xdir);
					radius = gradius[rotID];
					axis_x = gaxises[rotID * 3 + 0];
					axis_y = gaxises[rotID * 3 + 1];
					axis_z = gaxises[rotID * 3 + 2];
				}
				float3 midp = (ppos + ray + lpos) * 0.5f;      ////////////////////////////////////////////!!!!!!!!!!!!!!!!!!!!!!!!!! + ray
				unsigned int midx = unsigned int((dot3(midp, axis_x) / radius.x + 1)*res2);
				unsigned int midy = unsigned int((dot3(midp, axis_y) / radius.y + 1)*res2);

				if (midx < 0 || midx >= resolution || midy < 0 || midy >= resolution){ // line segment didnt intersect with scene.
					littedCnt++;
				}
				else{
					unsigned int z1 = unsigned int((dot3(ppos + ray, axis_z) / radius.z + 1)*res2); //////////////////////////////!!!!!!!!!!!!!!!!!!!+ray
					unsigned int z2 = unsigned int((dot3(lpos, axis_z) / radius.z + 1)*res2);
					if (z1>z2) { unsigned int tmp = z1; z1 = z2; z2 = tmp; }

					if (z1 < resolution && z2 >= 0){
						if (z1 < 0) z1 = 0;
						if (z2 >= resolution) z2 = resolution - 1;
						unsigned int * tptr = &(gtags[(rotID << (precision * 3 - 5)) + (midx << (precision * 2 - 5)) + (midy << (precision - 5))]);

						unsigned int zs = z1 >> 5;  unsigned int ze = z2 >> 5;
						unsigned int m1 = z1 & 31u; unsigned int m2 = z2 & 31u;
						if (zs == ze){ if (!((tptr[zs] << (32 - m2)) >> (32 - m2 + m1))) littedCnt++; }
						else{
							bool flag = false;
							for (int c = zs + 1; c < ze; c++) if (tptr[c]){ flag = true; break; };
							if (!flag){
								if ((!((~((1u << (m1 + 1)) - 1u))&(tptr[zs]))) && (!(((1u << (m2)) - 1u)&(tptr[ze])))){ littedCnt++; }
							}
						}
					}
					else littedCnt++;
				}
			}
		}
		ssbo_counter[y*width + x] += littedCnt;
	}
}

__global__ void compute_error(unsigned int* ours,unsigned int* optix,unsigned char* small_err/*,unsigned int* big_err*/,unsigned int* errs){
	const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (offset == 0){ errs[0] = errs[1] = 0; }
	__syncthreads();
	unsigned int* ours_ptr = &ours[offset*_UINT_CNT_PER_PIXEL_RESULT_];
	unsigned int* optix_ptr = &optix[offset*_UINT_CNT_PER_PIXEL_RESULT_];
	int our_cnt = 0;
	int optix_cnt = 0;
	int diff_cnt = 0;
	for (int i = 0; i < _UINT_CNT_PER_PIXEL_RESULT_; i++){
		diff_cnt += cnt1(ours_ptr[i] ^ optix_ptr[i]);
		our_cnt += cnt1(ours_ptr[i]);
		optix_cnt += cnt1(optix_ptr[i]);
	}
	atomicAdd(errs+0,diff_cnt);
	atomicAdd(errs+1,abs(our_cnt-optix_cnt));

	unsigned char* img_ptr = &small_err[offset * 3];
	if (our_cnt < optix_cnt){
		img_ptr[0] = ((optix_cnt - our_cnt) / 1024.0f * 10 * 255); if (img_ptr[0]>255) img_ptr[0] = 255;
		img_ptr[1] = img_ptr[2] = 0;
	}
	else{
		img_ptr[2] = (our_cnt - optix_cnt) / 1024.0f * 10 * 255; if (img_ptr[2]>255) img_ptr[2] = 255;
		img_ptr[0] = img_ptr[1] = 0;
	}
}

/**************************HOST FUNCTION*************************/
int lightCnt = 1024;
unsigned int scr_width = 512;
unsigned int  scr_height = 512;
unsigned int  rotcnt = 90;
unsigned int  precision = 7;
bool useCompressedColumns = false;
float3 boxcenter;

float3* lights_dev = nullptr;
float3* axises_dev = nullptr;
float3* radius_dev = nullptr;
unsigned int *tags_dev = nullptr;
unsigned int *indexes_dev = nullptr;
unsigned int *columns_dev = nullptr;
unsigned int *visibilities_dev = nullptr;
unsigned int *shadowMapAB_devptr = nullptr;

unsigned char *errorImage_dev = nullptr;
unsigned int *errors_dev = nullptr;

hipChannelFormatDesc channelDesc;
hipGraphicsResource *postex_resource, *normtex_resource, *shadowMap_resource;

extern "C" void blocker_initialize_cuda(GLuint postex,GLuint normtex,GLuint shadowmap,
	                                                      int lcnt,unsigned int width,unsigned int height,unsigned int rcnt,unsigned int columnCnt,unsigned int pres,bool compressed,redips::float3 heart,
														  const float* lights,const float* axises,const float* radius,const unsigned int* tags,const unsigned int* indexes,const unsigned int* columns)
{
	checkCudaErrors(hipGraphicsGLRegisterImage(&postex_resource, postex, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsGLRegisterImage(&normtex_resource, normtex, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&shadowMap_resource, shadowmap, cudaGraphicsMapFlagsWriteDiscard));
	
	channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	
	pos_ref.addressMode[0] = hipAddressModeWrap;
	pos_ref.addressMode[1] = hipAddressModeWrap;
	pos_ref.filterMode = hipFilterModeLinear;
	pos_ref.normalized = true;

	norm_ref.addressMode[0] = hipAddressModeWrap;
	norm_ref.addressMode[1] = hipAddressModeWrap;
	norm_ref.filterMode = hipFilterModeLinear;
	norm_ref.normalized = true;

	lightCnt = lcnt;
	scr_width = width;
	scr_height = height;
	rotcnt = rcnt;
	precision = pres;
	useCompressedColumns = compressed;
	boxcenter = make_float3(heart.x,heart.y,heart.z);

	//COMMON
	checkCudaErrors(hipMalloc((void**)&lights_dev, sizeof(float)* 3 * lightCnt));
	checkCudaErrors(hipMemcpy(lights_dev, lights, sizeof(float)* 3 * lightCnt, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&axises_dev, sizeof(float)* 9 * rotcnt*rotcnt));
	checkCudaErrors(hipMemcpy(axises_dev, axises, sizeof(float)* 9 * rotcnt*rotcnt, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&radius_dev, sizeof(float)* 3 * rotcnt *rotcnt));
	checkCudaErrors(hipMemcpy(radius_dev, radius, sizeof(float)* 3 * rotcnt*rotcnt, hipMemcpyHostToDevice));
	
	if (useCompressedColumns){
		checkCudaErrors(hipMalloc((void**)&columns_dev, (columnCnt << (precision - 5))*sizeof(unsigned int)));
		checkCudaErrors(hipMemcpy(columns_dev, columns, sizeof(unsigned int)*(columnCnt << (precision - 5)), hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void**)&indexes_dev, sizeof(unsigned int)*rotcnt*rotcnt*(1u << (precision * 2))));
		checkCudaErrors(hipMemcpy(indexes_dev, indexes, sizeof(unsigned int)*rotcnt*rotcnt*(1u << (precision * 2)), hipMemcpyHostToDevice));
	}
	else{
		checkCudaErrors(hipMalloc((void**)&tags_dev, (1u << (precision * 3 - 3))*rotcnt*rotcnt));
		checkCudaErrors(hipMemcpy(tags_dev, tags, (1u << (precision * 3 - 3))*rotcnt*rotcnt, hipMemcpyHostToDevice));
	}

	//for result
	checkCudaErrors(hipMalloc((void**)&visibilities_dev, sizeof(unsigned int)*_UINT_CNT_PER_PIXEL_RESULT_*scr_width*scr_height));

	//for error
	checkCudaErrors(hipMalloc((void**)&errorImage_dev,sizeof(unsigned char)*3*scr_width*scr_height));
	checkCudaErrors(hipMalloc((void**)&errors_dev, sizeof(unsigned int)* 2));
	/***********************delete********************/
	unsigned int* tmp = new unsigned int[512*512*32];
	std::ifstream fin("D:/��ѧ������-���꼴ɾ/AnalyseData/sponza/rtresult0");   
	for (int i = 0; i < 512*512*32; i++) {
		fin >> tmp[i];
	}
	fin.close();
	hipMemcpy(visibilities_dev,tmp,sizeof(unsigned int)*512*512*32,hipMemcpyHostToDevice);
	/***********************delete********************/
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" void launch_4_compare(){ //for compare
	hipArray *pos_array, *norm_array;
	//map position texture
	checkCudaErrors(hipGraphicsMapResources(1, &postex_resource, 0));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&pos_array, postex_resource, 0, 0));
	//map normal texture
	checkCudaErrors(hipGraphicsMapResources(1, &normtex_resource, 0));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&norm_array, normtex_resource, 0, 0));
	hipBindTextureToArray(pos_ref, pos_array, channelDesc);
	hipBindTextureToArray(norm_ref, norm_array, channelDesc);
	thrust::device_ptr<unsigned int > vis_ptr(visibilities_dev);       //may waste time
	thrust::fill(vis_ptr, vis_ptr + scr_width*scr_height*_UINT_CNT_PER_PIXEL_RESULT_, 0u);
	checkCudaErrors(hipDeviceSynchronize());

	dim3 blockDim(32, 32);
	dim3 gridDim((scr_width - 1) / 32 + 1, (scr_height - 1) / 32 + 1);
	for (int sid = 0; sid < lightCnt; sid += _MAX_LIGHT_CNT_PER_PROCESS_){
		int eid = sid + _MAX_LIGHT_CNT_PER_PROCESS_;  if (eid>lightCnt) eid = lightCnt;
		if (useCompressedColumns){
			calVisibility_compressed << <gridDim, blockDim >> >(scr_width, scr_height, eid - sid, sid, lights_dev + sid, rotcnt, axises_dev, radius_dev, columns_dev, indexes_dev, boxcenter, precision, visibilities_dev);
		}
		else{
			calVisibility_uncompressed << <gridDim, blockDim >> >(scr_width, scr_height, eid - sid, sid, lights_dev + sid, rotcnt, axises_dev, radius_dev, tags_dev, boxcenter, precision, visibilities_dev);
		}
		checkCudaErrors(hipDeviceSynchronize());
		if (eid >= lightCnt) break;
	}

	checkCudaErrors(hipGraphicsUnmapResources(1, &postex_resource, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &normtex_resource, 0));
}

extern "C" void launch_4_rendering(){ //for rendering
	//map position texture
	hipArray *pos_array, *norm_array;
	checkCudaErrors(hipGraphicsMapResources(1, &postex_resource, 0));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&pos_array, postex_resource, 0, 0));
	//map normal texture
	checkCudaErrors(hipGraphicsMapResources(1, &normtex_resource, 0));
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&norm_array, normtex_resource, 0, 0));
	hipBindTextureToArray(pos_ref, pos_array, channelDesc);
	hipBindTextureToArray(norm_ref, norm_array, channelDesc);
	//map shadow-map buffer
	size_t num_bytes;
	checkCudaErrors(hipGraphicsMapResources(1, &shadowMap_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&shadowMapAB_devptr, &num_bytes, shadowMap_resource));
	thrust::device_ptr<unsigned int > ssbo_ptr(shadowMapAB_devptr);       //may waste time
	thrust::fill(ssbo_ptr, ssbo_ptr + scr_width*scr_height, 0u);
	checkCudaErrors(hipDeviceSynchronize());

	dim3 blockDim(32, 32);
	dim3 gridDim((scr_width - 1) / 32 + 1, (scr_height - 1) / 32 + 1);

	float elapseTime; hipEvent_t start, stop;	hipEventCreate(&start);  hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int sid = 0; sid < lightCnt; sid += _MAX_LIGHT_CNT_PER_PROCESS_){
		int eid = sid + _MAX_LIGHT_CNT_PER_PROCESS_;  if (eid>lightCnt) eid = lightCnt;
		if (useCompressedColumns){
			genShadowMap_compressed << <gridDim, blockDim >> >(scr_width, scr_height, eid - sid, sid, lights_dev + sid, rotcnt, axises_dev, radius_dev, columns_dev, indexes_dev, boxcenter, precision, shadowMapAB_devptr);
		}
		else{
			genShadowMap_uncompressed << <gridDim, blockDim >> >(scr_width, scr_height, eid - sid, sid, lights_dev + sid, rotcnt, axises_dev, radius_dev, tags_dev, boxcenter, precision, shadowMapAB_devptr);
		}
		checkCudaErrors(hipDeviceSynchronize());
		if (eid >= lightCnt) break;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapseTime, start, stop);
	//printf("[blocker] :calculate visibility cost %.2f ms\n", elapseTime);

	checkCudaErrors(hipGraphicsUnmapResources(1, &postex_resource, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &normtex_resource, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &shadowMap_resource, 0));
}

hipGraphicsResource *optix_buffer_resource;
extern "C" void compute_errors(GLuint optix_buffer,unsigned char* img_cpu,redips::int2& ret){
	size_t num_bytes; unsigned int* optix_buffer_ptr;
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&optix_buffer_resource, optix_buffer, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsMapResources(1, &optix_buffer_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&optix_buffer_ptr, &num_bytes, optix_buffer_resource));

	compute_error<<<scr_height, scr_width >>>(visibilities_dev, optix_buffer_ptr, errorImage_dev, errors_dev);
	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(img_cpu,errorImage_dev,sizeof(unsigned int)*3*scr_width*scr_height,hipMemcpyDeviceToHost);
	hipMemcpy(&ret.x, errors_dev,sizeof(unsigned int)*2,hipMemcpyDeviceToHost);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGraphicsUnmapResources(1, &optix_buffer_resource, 0));
}

extern "C" void clean_cuda_blocker(){
	if (lights_dev) checkCudaErrors(hipFree(lights_dev));              lights_dev = nullptr;
	if (axises_dev) checkCudaErrors(hipFree(axises_dev));             axises_dev = nullptr;
	if (radius_dev) checkCudaErrors(hipFree(radius_dev));             radius_dev = nullptr;
	if (tags_dev) checkCudaErrors(hipFree(tags_dev));                  tags_dev = nullptr;
	if (indexes_dev) checkCudaErrors(hipFree(indexes_dev));        indexes_dev = nullptr;
	if (columns_dev) checkCudaErrors(hipFree(columns_dev));     columns_dev = nullptr;
	if (visibilities_dev) checkCudaErrors(hipFree(visibilities_dev));   visibilities_dev = nullptr;
	if (errorImage_dev) checkCudaErrors(hipFree(errorImage_dev)); errorImage_dev = nullptr;
	if (errors_dev) checkCudaErrors(hipFree(errors_dev)); errors_dev = nullptr;
}