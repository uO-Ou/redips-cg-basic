#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
using namespace optix;

struct PerRayData_shadow{
	float distance;
};

//2������
rtTextureSampler<float4, 2>  position_texture;
rtTextureSampler<float4, 2>  normal_texture;

//3��buffer
rtBuffer<float, 1> lights_buffer;
rtBuffer<unsigned int, 1> shadowMap_buffer;
rtBuffer<unsigned int, 1> visibilities_buffer;

rtDeclareVariable(uint, light_cnt, , );                                  //����ray��type
rtDeclareVariable(uint, scr_width, , );                                 //width
rtDeclareVariable(uint, shadow_ray_type, , );                     //����ray��type
rtDeclareVariable(float, scene_epsilon, , );                         //epsilon
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );    //launch_index  

rtDeclareVariable(rtObject, shadow_casters, , );    

rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );   //����ray��Я������
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );                     //�����������Ϊt_hit


RT_PROGRAM void any_hit_shadow(){
	prd_shadow.distance = t_hit;
	rtTerminateRay();
}

//������ߣ�����ɼ���
RT_PROGRAM void calculate_visibilities(){
	float3 ray_origin = make_float3(tex2D(position_texture, launch_index.x, launch_index.y));
	float3 pixel_normal = make_float3(tex2D(normal_texture, launch_index.x, launch_index.y));
	unsigned int offsets = launch_index.y*scr_width + launch_index.x;
	unsigned int uint_cpp = (light_cnt - 1) / 32 + 1;
	unsigned int littedCnt = 0;
	//if (launch_index.x != 222 || launch_index.y != 222) return;;
	if (!isnan(ray_origin.x)) {
		for (unsigned int lid = 0u; lid < light_cnt; lid++){
			//float3 light_pos = make_float3(tex1Dfetch(lights_texture, lid));
			float3 light_pos = *((float3 *)(&lights_buffer[lid*3]));
			float3 L = light_pos - ray_origin;
			if (dot(L, pixel_normal)>0.0f){
				PerRayData_shadow prd;
				prd.distance = -1;

				float dist = sqrtf(dot(L, L));
				float3 ray_direction = L / dist;
				optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, shadow_ray_type, scene_epsilon, dist);
				rtTrace(shadow_casters, ray, prd);
				if (prd.distance==-1){
					littedCnt++;
					visibilities_buffer[offsets*uint_cpp + lid / 32u] |= 1u << (lid & 31u);
				}
			}
		}
	}
	shadowMap_buffer[offsets] = littedCnt;
}

RT_PROGRAM void exception(){

}
