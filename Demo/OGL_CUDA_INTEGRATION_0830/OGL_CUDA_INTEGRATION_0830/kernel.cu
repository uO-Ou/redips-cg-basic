#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <helper_gl.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <vec.h>

//**************************kernels
__global__ void calcnt(unsigned int *input,unsigned int *output,int workload){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (; tid < workload;tid+=gap){
		unsigned int status = input[tid];
		int cnt = 0;
		for (int i = 0; i < 32; i++){
			if (!status) break;
			if (status & 1u) cnt++;
			status >>= 1;
		}
		output[tid] = cnt;
	}
}
__global__ void gendots(unsigned int *input, unsigned int *presum, float3* dots, int res2, int workload, redips::float3 boxcenter, redips::float3 boxdim){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float resolution = 1u << res2;
	float3 dot;
	for (; tid < workload; tid += gap){
		unsigned int status = input[tid];
		if (!status) continue;
		float3* ptrf3 = dots + presum[tid];
		unsigned int voxelx = tid >> (res2+res2 - 5);
		unsigned int voxely = (tid&((1u<<(res2+res2-5))-1u)) >> (res2 - 5);
		unsigned int voxelz = (tid&((1u<<(res2 - 5)) - 1)) << 5;
		
		for (int i = 0; i < 32; i++){
			if (status&(1u << i)) {
				dot.x = (voxelx / resolution - 0.5f)*boxdim.x + boxcenter.x;
				dot.y = (voxely / resolution - 0.5f)*boxdim.y + boxcenter.y;
				dot.z = ((voxelz + i) / resolution - 0.5f)*boxdim.z + boxcenter.z;
				*(ptrf3++) = dot;
			}
		}
	}
}


extern "C"
bool cudaInit(){
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}
	return true;
}

extern "C"
GLuint packVoxel(GLuint invbo, int res2, unsigned int &TOTAL_VOXEL_CNT,redips::float3 boxcenter, redips::float3 boxdim){
	GLuint outvbo = 0;
	unsigned int* inputptr;  float3* outptr; size_t num_bytes;
	struct hipGraphicsResource *cuda_input,*cuda_output;
	//step1: map vbo 2 cuda
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_input, invbo, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_input, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&inputptr, &num_bytes, cuda_input));
	SDK_CHECK_ERROR_GL();
	printf("[cuda] : mapped %d bytes voxel input\n", num_bytes);
	//step2: compact, calculate presum
	unsigned int workload = 1u << (res2*3-5);
	unsigned int* presum_dev;
	checkCudaErrors(hipMalloc((void**)&presum_dev,workload*sizeof(unsigned int)));
	calcnt << <(workload - 1) / 512 + 1, 512 >> >(inputptr,presum_dev,workload);
	hipDeviceSynchronize();
	thrust::device_ptr<unsigned int > presumPtr(presum_dev);
	thrust::exclusive_scan(presumPtr, presumPtr + workload, presumPtr);
	hipDeviceSynchronize();
	hipMemcpy(&TOTAL_VOXEL_CNT,presum_dev+workload-1,sizeof(unsigned int),hipMemcpyDeviceToHost);
	printf("[cuda] : total voxel cnt is %d\n",TOTAL_VOXEL_CNT);
	//step3: new buffer,generate dots 
	glGenBuffers(1,&outvbo);
	glBindBuffer(GL_ARRAY_BUFFER, outvbo);
	glBufferData(GL_ARRAY_BUFFER, TOTAL_VOXEL_CNT*sizeof(float)* 3, nullptr, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_output, outvbo, cudaGraphicsMapFlagsWriteDiscard));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&outptr, &num_bytes, cuda_output));
	printf("[cuda] : mapped %d bytes voxel out\n", num_bytes);
	gendots << <(workload - 1) / 512 + 1, 512 >> >(inputptr, presum_dev, outptr, res2, workload,boxcenter,boxdim);
	
	//step4. unmap, release memory
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_input, 0));
	hipFree(presum_dev);
	return outvbo;
}

