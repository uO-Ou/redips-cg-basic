#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <helper_gl.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <vec.h>

#define _MAX_GMEM_4_VOXELRESULT_IN_GB_ 2.0
#define _DIRECTION_CNT_PER_BLOCK_ 16u
#define _THREAD_NUM_PER_BLOCK_ 1024

/******************************************************************/
/**********                          kernels                               ***********/
/******************************************************************/
__device__ void multy(){

}
__global__ void calcnt(unsigned int *input,unsigned int *output,int workload){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (; tid < workload;tid+=gap){
		unsigned int status = input[tid];
		int cnt = 0;
		for (int i = 0; i < 32; i++){
			if (!status) break;
			if (status & 1u) cnt++;
			status >>= 1;
		}
		output[tid] = cnt;
	}
}
__global__ void gendots(unsigned int *input, unsigned int *presum, float3* dots, int res2, int workload, redips::float3 boxcenter, redips::float3 boxdim){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float resolution = 1u << res2;
	float3 dot;
	for (; tid < workload; tid += gap){
		unsigned int status = input[tid];
		if (!status) continue;
		float3* ptrf3 = dots + presum[tid];
		unsigned int voxelx = tid >> (res2+res2 - 5);
		unsigned int voxely = (tid&((1u<<(res2+res2-5))-1u)) >> (res2 - 5);
		unsigned int voxelz = (tid&((1u<<(res2 - 5)) - 1)) << 5;
		
		for (int i = 0; i < 32; i++){
			if (status&(1u << i)) {
				dot.x = (voxelx / resolution - 0.5f)*boxdim.x + boxcenter.x;
				dot.y = (voxely / resolution - 0.5f)*boxdim.y + boxcenter.y;
				dot.z = ((voxelz + i) / resolution - 0.5f)*boxdim.z + boxcenter.z;
				*(ptrf3++) = dot;
			}
		}
	}
}
__global__ void transform(float3* dots, unsigned dotcnt, redips::float3 center, float* mats,unsigned int sid,unsigned int eid,unsigned int presicion,unsigned int * result){
	const unsigned int gap = blockDim.x;
	unsigned int tid = threadIdx.x;
	unsigned int matSId = blockIdx.x * _DIRECTION_CNT_PER_BLOCK_;
	unsigned int matEId = matSId + _DIRECTION_CNT_PER_BLOCK_ - 1; if (matEId + sid > eid) matEId = eid - sid;

	__shared__ float mats_sm[_DIRECTION_CNT_PER_BLOCK_ << 4];
	__shared__ float dots_sm[_THREAD_NUM_PER_BLOCK_*3];
	{
		float* matsptr = mats + ((sid+matSId) << 4);
		if (tid < ((matEId - matSId + 1) << 4)){
			mats_sm[tid] = matsptr[tid];
		}
		__syncthreads();
	}
	
	for (; tid < dotcnt; tid += gap){
		for (int mid = matSId; mid <= matEId; mid++){  // mid+sid 

		}
	}
}

extern "C"
bool cudaInit(){
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}
	return true;
}

extern "C"
GLuint packVoxel(GLuint invbo, int res2, unsigned int &TOTAL_VOXEL_CNT,redips::float3 boxcenter, redips::float3 boxdim){
	GLuint outvbo = 0;
	unsigned int* inputptr;  float3* outptr; size_t num_bytes;
	struct hipGraphicsResource *cuda_input,*cuda_output;
	//step1: map vbo 2 cuda
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_input, invbo, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_input, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&inputptr, &num_bytes, cuda_input));
	SDK_CHECK_ERROR_GL();
	printf("[cuda] : mapped %d bytes voxel input\n", num_bytes);
	//step2: compact, calculate presum
	unsigned int workload = 1u << (res2*3-5);
	unsigned int* presum_dev;
	checkCudaErrors(hipMalloc((void**)&presum_dev,workload*sizeof(unsigned int)));
	calcnt << <(workload - 1) / 512 + 1, 512 >> >(inputptr,presum_dev,workload);
	hipDeviceSynchronize();
	thrust::device_ptr<unsigned int > presumPtr(presum_dev);
	thrust::exclusive_scan(presumPtr, presumPtr + workload, presumPtr);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(&TOTAL_VOXEL_CNT, presum_dev + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));
	printf("[cuda] : total voxel cnt is %d\n",TOTAL_VOXEL_CNT);
	//step3: new buffer,generate dots 
	glGenBuffers(1,&outvbo);
	glBindBuffer(GL_ARRAY_BUFFER, outvbo);
	glBufferData(GL_ARRAY_BUFFER, TOTAL_VOXEL_CNT*sizeof(float)* 3, nullptr, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_output, outvbo, cudaGraphicsMapFlagsWriteDiscard));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&outptr, &num_bytes, cuda_output));
	printf("[cuda] : mapped %d bytes voxel out\n", num_bytes);
	gendots << <(workload - 1) / 512 + 1, 512 >> >(inputptr, presum_dev, outptr, res2, workload,boxcenter,boxdim);
	
	//step4. unmap, release memory
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_input, 0));
	hipFree(presum_dev);
	return outvbo;
}

extern "C"
void mdVoxelization(GLuint dotsvbo, int dotscnt, redips::float3 dotscenter, int rotx, int roty,const float* mats, unsigned int presicion){
	   float MEM_PER_DIRECTION_MB = (1u << (presicion * 3 - 3))*1.0f / (1u << 20);
	   size_t DIRECTION_CNT_PER_PROCESS = _MAX_GMEM_4_VOXELRESULT_IN_GB_*1024.0f / MEM_PER_DIRECTION_MB;
	   printf("[cuda] : MEM_PER_DIRECTION_MB is %.4f, DIRECTION_CNT_PER_PROCESS %lld using %.4fg memory\n", MEM_PER_DIRECTION_MB, DIRECTION_CNT_PER_PROCESS, DIRECTION_CNT_PER_PROCESS*MEM_PER_DIRECTION_MB/1024);
	   
	   float* dots_dev; size_t num_bytes;
	   struct hipGraphicsResource *cuda_vbo_binder;
	   checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_binder, dotsvbo, cudaGraphicsMapFlagsReadOnly));
	   checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_binder, 0));
	   checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dots_dev, &num_bytes, cuda_vbo_binder));
	   printf("[cuda] : mapped %d bytes dots\n", num_bytes);
	   
	   float* mats_dev;
	   checkCudaErrors(hipMalloc((void**)&mats_dev, sizeof(float)* 16 * rotx*roty));
	   checkCudaErrors(hipMemcpy(mats_dev, mats, sizeof(float)* 16 * rotx*roty, hipMemcpyHostToDevice));

	   unsigned int* result_dev;
	   checkCudaErrors(hipMalloc((void**)&result_dev, DIRECTION_CNT_PER_PROCESS*(1u << (presicion * 3 - 3))));

	   //launch kernels
	   for (int sid = 0; ;sid += DIRECTION_CNT_PER_PROCESS){
		   int tid = MIN(sid + DIRECTION_CNT_PER_PROCESS - 1, rotx*roty - 1);
		   int blockCnt = ((tid - sid + 1) - 1) / _DIRECTION_CNT_PER_BLOCK_ + 1;

		   if (tid >= rotx*roty - 1) break;
	   }

	   checkCudaErrors(hipFree(result_dev));
	   checkCudaErrors(hipFree(mats_dev));
	   checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_binder, 0));
}