#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <direct.h>
#include <helper_gl.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <vec.h>

#define _MAX_GMEM_4_VOXELRESULT_IN_GB_ 2.0f
#define _DIRECTION_CNT_PER_BLOCK_ 32u
#define _THREAD_NUM_PER_BLOCK_ 512
#define CC(x) {const hipError_t a = (x); if(a!=hipSuccess){printf("\ncuda error:%s(err_num=%d)\n",hipGetErrorString(a),a);hipDeviceReset();assert(0);}}

/******************************************************************/
/**********                    device function                        ***********/
/******************************************************************/
__device__ float3 multy(float* mat,const float4& dot){
	float3 ret;
	float winv = 1.0f/((mat[12] * dot.x) + (mat[13] * dot.y) + (mat[14] * dot.z) + mat[15]);
	ret.x = ((mat[0] * dot.x) + (mat[1] * dot.y) + (mat[2] * dot.z) + mat[3])*winv;
	ret.y = ((mat[4] * dot.x) + (mat[5] * dot.y) + (mat[6] * dot.z) + mat[7])*winv;
	ret.z = ((mat[8] * dot.x) + (mat[9] * dot.y) + (mat[10] * dot.z) + mat[11])*winv;
	return ret;
}
__inline__ __device__ unsigned int  mclamp(unsigned int a, unsigned int b, unsigned c){
	if (a <= b) return b;
	if (a >= c) return c;
	return a;
}
/*__device__ unsigned char atomicOrChar(unsigned char* address, unsigned char val){
	unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
	unsigned int selectors[] = { 0x3214, 0x3240, 0x3410, 0x4210 };
	unsigned int sel = selectors[(size_t)address & 3];
	unsigned int old, assumed, min_, new_;
	old = *base_address;
	do {
		assumed = old;
		min_ = min(val, (char)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
		new_ = __byte_perm(old, min_, sel);
		if (new_ == old)
			break;
		old = atomicCAS(base_address, assumed, new_);
	} while (assumed != old);
	return old;
}*/


/******************************************************************/
/**********                          kernels                               ***********/
/******************************************************************/
//allocate frags to cell
__global__ void frags2Cell(float4* frags, int fragcnt, redips::float3 center, float *mats, unsigned int precision, unsigned int * result){
	const int gap = blockDim.x * gridDim.x;
	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ float mats_sm[16];
	if (threadIdx.x < 16) mats_sm[threadIdx.x] = mats[threadIdx.x];
	__syncthreads();

	unsigned int resolution = 1u << precision;
	unsigned int res2 = resolution >> 1u;
	for (int fid = tid; fid < fragcnt; fid += gap){
		float4 tdot = frags[fid];   tdot.x -= center.x, tdot.y -= center.y, tdot.z -= center.z;
		float3 ndc = multy(mats_sm, tdot);

		unsigned int tx = 0.5f + ndc.x * res2 + res2;  tx = mclamp(tx, 0, resolution - 1);
		unsigned int ty = 0.5f + ndc.y * res2 + res2;  ty = mclamp(ty, 0, resolution - 1);
		unsigned int tz = 0.5f + ndc.z * res2 + res2;  tz = mclamp(tz, 0, resolution - 1);

		atomicOr(&result[(tx << (precision * 2 - 5)) + (ty << (precision - 5)) + (tz >> 5)], 1u << (tz & 31u));
	}
}
//calculate 1's count in each cell
__global__ void calcnt(unsigned int *input,unsigned int *output,int workload){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (; tid < workload;tid+=gap){
		unsigned int status = input[tid];
		int cnt = 0;
		for (int i = 0; i < 32; i++){
			if (!status) break;
			if (status & 1u) cnt++;
			status >>= 1;
		}
		output[tid] = cnt;
	}
}
//calculate cell center's coordinate
__global__ void gendots(unsigned int *input, unsigned int *presum, float3* output, float* axis,int precision, int workload, redips::float3 boxcenter, redips::float3 boxdim){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float resolution = 1u << precision;
	float3 dot;
	for (; tid < workload; tid += gap){
		unsigned int status = input[tid];
		if (!status) continue;
		float3* ptrf3 = output + presum[tid];
		unsigned int voxelx = tid >> (precision + precision - 5);
		unsigned int voxely = (tid&((1u << (precision + precision - 5)) - 1u)) >> (precision - 5);
		unsigned int voxelz = (tid&((1u << (precision - 5)) - 1u)) << 5;
		
		for (int i = 0; i < 32; i++){
			if (status&(1u << i)) {
			    float tx = (voxelx / resolution - 0.5f)*boxdim.x;
			    float ty = (voxely / resolution - 0.5f)*boxdim.y;
				float tz = ((voxelz + i) / resolution - 0.5f)*boxdim.z;

				dot.x = tx * axis[0] + ty * axis[3] + tz * axis[6] + boxcenter.x;
				dot.y = tx * axis[1] + ty * axis[4] + tz * axis[7] + boxcenter.y;
				dot.z = tx * axis[2] + ty * axis[5] + tz * axis[8] + boxcenter.z;

				*(ptrf3++) = dot;
			}
		}
	}
}
//transform frags [sid-eid] to cell
__global__ void transform(float4* frags, int fragcnt, redips::float3 center, float* mats,unsigned int sid,unsigned int eid,unsigned int precision,unsigned int * result){
	unsigned int tid = threadIdx.x;
	unsigned int matSId = blockIdx.x * _DIRECTION_CNT_PER_BLOCK_;
	unsigned int matEId = matSId + _DIRECTION_CNT_PER_BLOCK_ - 1; if (matEId + sid > eid) matEId = eid - sid;
	unsigned int matCnt = matEId - matSId + 1;
	unsigned int resolution = 1u << precision;
	unsigned int shiftleft = precision * 3 - 5;            //*
	unsigned int *resultPtr = &result[(matSId<<shiftleft)]; //*

	__shared__ float mats_sm[_DIRECTION_CNT_PER_BLOCK_ << 4];
	{
		float* matsptr = mats + ((sid+matSId) << 4);
		if (tid < (matCnt << 4)){
			mats_sm[tid] = matsptr[tid];
		}
		__syncthreads();
	}
	
	unsigned int res2 = resolution >> 1;
	for (; tid < fragcnt; tid += _THREAD_NUM_PER_BLOCK_){
		float4 dot = frags[tid]; 
		dot.x -= center.x; dot.y -= center.y; dot.z -= center.z;
		for (unsigned int mid = 0; mid < matCnt; mid++){  // mid+sid 
			float3 ndc = multy(&(mats_sm[(mid<<4)]),dot);
			
			unsigned int tx = 0.5f + ndc.x * res2 + res2;  tx = mclamp(tx, 0, resolution - 1);
			unsigned int ty = 0.5f + ndc.y * res2 + res2;  ty = mclamp(ty, 0, resolution - 1);
			unsigned int tz = 0.5f + ndc.z * res2 + res2;  tz = mclamp(tz, 0, resolution - 1);

			unsigned int offset = (mid<<shiftleft) + ((tx<<(precision*2 - 5)) + (ty<<(precision - 5)) + (tz>>5));

			atomicOr(&resultPtr[offset], (1u << (tz & 31u)));
		}
	}
}


/******************************************************************/
/**********                    host function                        ***********/
/******************************************************************/
extern "C"
bool cudaInit(){
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}
	return true;
}

extern "C"
unsigned int compact_cuda(GLuint invbo,int precision){
	unsigned int* inputptr; size_t num_bytes;
	struct hipGraphicsResource *cuda_input;
	//map vbo 2 cuda
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_input, invbo, cudaGraphicsMapFlagsNone));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_input, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&inputptr, &num_bytes, cuda_input));
	SDK_CHECK_ERROR_GL();
	printf("[cuda] : mapped %d bytes voxel input\n", num_bytes);

	unsigned int workload = (1u << (precision * 3 - 5));

	unsigned int LAST_CELL_CNT = 0;
	checkCudaErrors(hipMemcpy(&LAST_CELL_CNT, inputptr + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));
	CC(hipDeviceSynchronize());

	thrust::device_ptr<unsigned int> presumPtr(inputptr);
	thrust::exclusive_scan(presumPtr, presumPtr + workload, presumPtr);
	CC(hipDeviceSynchronize());

	unsigned int TOTAL_FRAG_CNT = 0;
	checkCudaErrors(hipMemcpy(&TOTAL_FRAG_CNT, inputptr + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));

	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_input, 0));
	return TOTAL_FRAG_CNT + LAST_CELL_CNT;
};

extern "C"
unsigned int onedVoxelization_cuda(GLuint fragsvbo, unsigned int fragscnt, redips::float3 fragscenter, redips::float3 boxdim,const float* mats,const float* axises, unsigned int precision, GLuint resultvbo){
	float* frags_dev;  float3* voxels_dev; size_t num_bytes;
	struct hipGraphicsResource *cuda_vbo_binder1, *cuda_vbo_binder2;
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_binder1, fragsvbo, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_binder1, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&frags_dev, &num_bytes, cuda_vbo_binder1));
	printf("[cuda] : mapped %d bytes[%.fM] frags_ssbo\n", num_bytes, num_bytes*1.0f / 1024 / 1024);

	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_binder2, resultvbo, cudaGraphicsMapFlagsWriteDiscard));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_binder2, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&voxels_dev, &num_bytes, cuda_vbo_binder2));
	printf("[cuda] : mapped %d bytes[%.fM] voxel_vbo\n", num_bytes, num_bytes*1.0f / 1024 / 1024);

	float* mats_dev;
	CC(hipMalloc((void**)&mats_dev, sizeof(float) * 16));
	CC(hipMemcpy(mats_dev, mats, sizeof(float) * 16, hipMemcpyHostToDevice));

	unsigned int * cellflag_dev;
	CC(hipMalloc((void**)&cellflag_dev, (1u << (precision * 3 - 3))));
	CC(hipMemset(cellflag_dev, 0, (1u << (precision * 3 - 3))));
	CC(hipDeviceSynchronize());

	//allocate to cell
	(frags2Cell << <(fragscnt - 1) / 512 + 1, 512 >> >((float4*)frags_dev, fragscnt, fragscenter, mats_dev, precision, cellflag_dev));
	CC(hipDeviceSynchronize());

	//calculate presum
	unsigned int workload = 1u << (precision * 3 - 5);
	unsigned int * presum_dev;
	CC(hipMalloc((void**)&presum_dev, (1u << (precision * 3 - 3))));
	calcnt << <(workload - 1) / 512 + 1, 512 >> >(cellflag_dev, presum_dev, workload);
	CC(hipDeviceSynchronize());
	unsigned int LAST_CNT;
	checkCudaErrors(hipMemcpy(&LAST_CNT, presum_dev + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));

	thrust::device_ptr<unsigned int> presumPtr(presum_dev);
	thrust::exclusive_scan(presumPtr, presumPtr + workload, presumPtr);
	CC(hipDeviceSynchronize());
	unsigned int TOTAL_VOXEL_CNT;
	checkCudaErrors(hipMemcpy(&TOTAL_VOXEL_CNT, presum_dev + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));
	CC(hipDeviceSynchronize());

	//generate dots
	CC(hipMemcpy(mats_dev, axises, sizeof(float)* 9, hipMemcpyHostToDevice));
	gendots << <(workload - 1) / 512 + 1, 512 >> >(cellflag_dev, presum_dev, voxels_dev, mats_dev, precision, workload, fragscenter, boxdim);


	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_binder1, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_binder2, 0));
	checkCudaErrors(hipFree(mats_dev));
	checkCudaErrors(hipFree(cellflag_dev));
	checkCudaErrors(hipFree(presum_dev));
	return TOTAL_VOXEL_CNT + LAST_CNT;
}

extern "C"
void mdVoxelization_cuda(GLuint fragsvbo, unsigned int fragscnt, redips::float3 fragscenter, int rotx, int roty, const float* mats, unsigned int presicion, std::string outputdir){
	   //prepare
	   float MEM_PER_DIRECTION_MB = (1u << (presicion * 3 - 3))*1.0f / (1u << 20);
	   int DIRECTION_CNT_PER_PROCESS = _MAX_GMEM_4_VOXELRESULT_IN_GB_ * 1024.0f / MEM_PER_DIRECTION_MB;
	   size_t result_uint_cnt = DIRECTION_CNT_PER_PROCESS*(1u << (presicion * 3 - 5));
	   printf("[cuda] : MEM_PER_DIRECTION_MB is %.4f, DIRECTION_CNT_PER_PROCESS %lld using %.4f g memory\n", MEM_PER_DIRECTION_MB, DIRECTION_CNT_PER_PROCESS, DIRECTION_CNT_PER_PROCESS*MEM_PER_DIRECTION_MB/1024.0f);
	   unsigned int * rbuffer = new unsigned int[result_uint_cnt];
	   char strBuffer[256];  sprintf(strBuffer, "%s/%d", outputdir.c_str(), (1u << presicion)); outputdir = std::string(strBuffer)+"/";
	   _mkdir(strBuffer);
	   
	   //allocate memory
	   float* frags_dev; size_t num_bytes;
	   struct hipGraphicsResource *cuda_vbo_binder;
	   checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_binder, fragsvbo, cudaGraphicsMapFlagsReadOnly));
	   checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_binder, 0));
	   checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&frags_dev, &num_bytes, cuda_vbo_binder));
	   printf("[cuda] : mapped %d bytes[%.fM] dots\n", num_bytes,num_bytes*1.0f/1024/1024);
	   
	   float* mats_dev;
	   checkCudaErrors(hipMalloc((void**)&mats_dev, sizeof(float)* 16 * rotx * roty));
	   checkCudaErrors(hipMemcpy(mats_dev, mats, sizeof(float)* 16 * rotx * roty, hipMemcpyHostToDevice));

	   unsigned int* result_dev;
	   checkCudaErrors(hipMalloc((void**)&result_dev, result_uint_cnt*sizeof(unsigned int)));
	   thrust::device_ptr<unsigned int > resultSPtr(result_dev);
	   thrust::device_ptr<unsigned int > resultEPtr(result_dev + result_uint_cnt);
	   
	   //launch kernels
	   int cid = 0;
	   StopWatchInterface *timer = 0;
	   sdkCreateTimer(&timer);
	   hipDeviceSynchronize();
	   sdkStartTimer(&timer);
	   
	   for (int sid = 0; ;sid += DIRECTION_CNT_PER_PROCESS){
		   int tid = std::min(sid + DIRECTION_CNT_PER_PROCESS - 1, rotx*roty - 1);
		   int blockCnt = ((tid - sid + 1) - 1) / _DIRECTION_CNT_PER_BLOCK_ + 1;

		   printf("[cuda] : dealing %d - %d\n",sid,tid);
		   thrust::fill(resultSPtr, resultEPtr,0u);
		   CC(hipDeviceSynchronize());

		   transform << <blockCnt, _THREAD_NUM_PER_BLOCK_ >> >((float4*)frags_dev, fragscnt, fragscenter, mats_dev, sid, tid, presicion, result_dev);
		   CC(hipDeviceSynchronize());

		   printf("[cuda] : copying to cpu \n");
		   checkCudaErrors(hipMemcpy(rbuffer, result_dev, size_t(tid - sid + 1)*(1u<<(presicion*3-3)),hipMemcpyDeviceToHost));
		   CC(hipDeviceSynchronize());
		   
		   printf("[cuda] : writing to hard disk \n");
		   unsigned int *uintptr = rbuffer;
		   unsigned int cnt_per_line = (1u << (presicion - 5));
		   for (int id = sid; id <= tid; id++){
			   int anglex = id % rotx;
			   int angley = id / rotx;
			   sprintf(strBuffer, "x%02d_y%02d.txt", anglex, angley);

			   freopen((outputdir+strBuffer).c_str(),"w",stdout);
			   for (unsigned int ind = 0; ind < (1u << (presicion * 3 - 5)); ind+=cnt_per_line){
				   for (unsigned int step = 0; step < cnt_per_line; step++) printf("%u ",uintptr[ind+step]); puts("");
			   }
			   fclose(stdout);

			   uintptr += (1u << (presicion * 3 - 5));
		   }
		   freopen("CON","w",stdout);
		   
		   printf("[cuda] : iteration %d finish \n", cid++);
		   if (tid >= rotx*roty - 1) break;
	   }
	   sdkStopTimer(&timer);

	   float elapseTime = 1.0e-3 * sdkGetTimerValue(&timer);
	   printf("[cuda] : transform total cost %f s\n",elapseTime);

	   delete rbuffer;
	   sdkDeleteTimer(&timer);
	   checkCudaErrors(hipFree(result_dev));
	   checkCudaErrors(hipFree(mats_dev));
	   checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_binder, 0));
}

