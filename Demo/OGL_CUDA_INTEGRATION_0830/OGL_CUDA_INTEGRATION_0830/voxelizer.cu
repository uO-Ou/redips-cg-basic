#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <direct.h>
#include <helper_gl.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <vec.h>

#define _MAX_GMEM_4_VOXELRESULT_IN_GB_ 2.0
#define _DIRECTION_CNT_PER_BLOCK_ 16u
#define _THREAD_NUM_PER_BLOCK_ 1024

/******************************************************************/
/**********                          kernels                               ***********/
/******************************************************************/
__inline__ __device__ float3 multy(float* mat,const float3& dot){
	float3 ret;
	float winv = 1.0f/((mat[12] * dot.x) + (mat[13] * dot.y) + (mat[14] * dot.z) + mat[15]);
	ret.x = ((mat[0] * dot.x) + (mat[1] * dot.y) + (mat[2] * dot.z) + mat[3])*winv;
	ret.y = ((mat[4] * dot.x) + (mat[5] * dot.y) + (mat[6] * dot.z) + mat[7])*winv;
	ret.z = ((mat[8] * dot.x) + (mat[9] * dot.y) + (mat[10] * dot.z) + mat[11])*winv;
	return ret;
}

/*__device__ unsigned char atomicOrChar(unsigned char* address, unsigned char val){
	unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
	unsigned int selectors[] = { 0x3214, 0x3240, 0x3410, 0x4210 };
	unsigned int sel = selectors[(size_t)address & 3];
	unsigned int old, assumed, min_, new_;
	old = *base_address;
	do {
		assumed = old;
		min_ = min(val, (char)__byte_perm(old, 0, ((size_t)address & 3) | 0x4440));
		new_ = __byte_perm(old, min_, sel);
		if (new_ == old)
			break;
		old = atomicCAS(base_address, assumed, new_);
	} while (assumed != old);
	return old;
}*/

__global__ void calcnt(unsigned int *input,unsigned int *output,int workload){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (; tid < workload;tid+=gap){
		unsigned int status = input[tid];
		int cnt = 0;
		for (int i = 0; i < 32; i++){
			if (!status) break;
			if (status & 1u) cnt++;
			status >>= 1;
		}
		output[tid] = cnt;
	}
}
__global__ void gendots(unsigned int *input, unsigned int *presum, float3* dots, int res2, int workload, redips::float3 boxcenter, redips::float3 boxdim){
	const unsigned int gap = gridDim.x * blockDim.x;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float resolution = 1u << res2;
	float3 dot;
	for (; tid < workload; tid += gap){
		unsigned int status = input[tid];
		if (!status) continue;
		float3* ptrf3 = dots + presum[tid];
		unsigned int voxelx = tid >> (res2+res2 - 5);
		unsigned int voxely = (tid&((1u<<(res2+res2-5))-1u)) >> (res2 - 5);
		unsigned int voxelz = (tid&((1u<<(res2 - 5)) - 1)) << 5;
		
		for (int i = 0; i < 32; i++){
			if (status&(1u << i)) {
				dot.x = (voxelx / resolution - 0.5f)*boxdim.x + boxcenter.x;
				dot.y = (voxely / resolution - 0.5f)*boxdim.y + boxcenter.y;
				dot.z = ((voxelz + i) / resolution - 0.5f)*boxdim.z + boxcenter.z;
				*(ptrf3++) = dot;
			}
		}
	}
}
__global__ void transform(float3* dots, unsigned dotcnt, redips::float3 center, float* mats,unsigned int sid,unsigned int eid,unsigned int precision,unsigned int * result){
	const unsigned int gap = blockDim.x;
	unsigned int tid = threadIdx.x;
	unsigned int matSId = blockIdx.x * _DIRECTION_CNT_PER_BLOCK_;
	unsigned int matEId = matSId + _DIRECTION_CNT_PER_BLOCK_ - 1; if (matEId + sid > eid) matEId = eid - sid;
	unsigned int matCnt = matEId - matSId + 1;
	unsigned int resolution = 1u << precision;
	unsigned int shiftleft = precision * 3 - 5;            //*
	unsigned int * resultPtr = result + (matSId<<shiftleft); //*

	__shared__ float mats_sm[_DIRECTION_CNT_PER_BLOCK_ << 4];
	{
		float* matsptr = mats + ((sid+matSId) << 4);
		if (tid < (matCnt << 4)){
			mats_sm[tid] = matsptr[tid];
		}
		__syncthreads();
	}
	
	
	unsigned int res2 = resolution >> 1;
	for (; tid < dotcnt; tid += gap){
		float3 dot = dots[tid]; 
		dot.x -= center.x; dot.y -= center.y; dot.z -= center.z;
		for (unsigned int mid = 0; mid < matCnt; mid++){  // mid+sid 
			float3 ndc = multy(mats_sm+(mid<<4),dot);
			unsigned int tx = 0.5f + ndc.x * res2 + res2;  tx = CLAMP(tx, 0, resolution - 1);
			unsigned int ty = 0.5f + ndc.y * res2 + res2;  ty = CLAMP(ty, 0, resolution - 1);
			unsigned int tz = 0.5f + ndc.z * res2 + res2;  tz = CLAMP(tz, 0, resolution - 1);
			
			atomicOr((resultPtr + (mid << shiftleft)) + ((tx << ((precision << 1) - 5)) + (ty << (precision - 5)) + (tz >> 5)),1u<<(tz&31u));
			
		}
	}
}

extern "C"
bool cudaInit(){
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}
	return true;
}

extern "C"
GLuint packVoxel(GLuint invbo, int res2, unsigned int &TOTAL_VOXEL_CNT,redips::float3 boxcenter, redips::float3 boxdim){
	GLuint outvbo = 0;
	unsigned int* inputptr;  float3* outptr; size_t num_bytes;
	struct hipGraphicsResource *cuda_input,*cuda_output;
	//step1: map vbo 2 cuda
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_input, invbo, cudaGraphicsMapFlagsReadOnly));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_input, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&inputptr, &num_bytes, cuda_input));
	SDK_CHECK_ERROR_GL();
	printf("[cuda] : mapped %d bytes voxel input\n", num_bytes);
	//step2: compact, calculate presum
	unsigned int workload = 1u << (res2*3-5);
	unsigned int* presum_dev;
	checkCudaErrors(hipMalloc((void**)&presum_dev,workload*sizeof(unsigned int)));
	calcnt << <(workload - 1) / 512 + 1, 512 >> >(inputptr,presum_dev,workload);
	hipDeviceSynchronize();
	thrust::device_ptr<unsigned int > presumPtr(presum_dev);
	thrust::exclusive_scan(presumPtr, presumPtr + workload, presumPtr);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(&TOTAL_VOXEL_CNT, presum_dev + workload - 1, sizeof(unsigned int), hipMemcpyDeviceToHost));
	printf("[cuda] : total voxel cnt is %d\n",TOTAL_VOXEL_CNT);
	//step3: new buffer,generate dots 
	glGenBuffers(1,&outvbo);
	glBindBuffer(GL_ARRAY_BUFFER, outvbo);
	glBufferData(GL_ARRAY_BUFFER, TOTAL_VOXEL_CNT*sizeof(float)* 3, nullptr, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_output, outvbo, cudaGraphicsMapFlagsWriteDiscard));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&outptr, &num_bytes, cuda_output));
	printf("[cuda] : mapped %d bytes voxel out\n", num_bytes);
	gendots << <(workload - 1) / 512 + 1, 512 >> >(inputptr, presum_dev, outptr, res2, workload,boxcenter,boxdim);
	
	//step4. unmap, release memory
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_output, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_input, 0));
	hipFree(presum_dev);
	return outvbo;
}

extern "C"
void mdVoxelization(GLuint dotsvbo, int dotscnt, redips::float3 dotscenter, int rotx, int roty,const float* mats, unsigned int presicion,std::string outputdir){
	   float MEM_PER_DIRECTION_MB = (1u << (presicion * 3 - 3))*1.0f / (1u << 20);
	   size_t DIRECTION_CNT_PER_PROCESS = _MAX_GMEM_4_VOXELRESULT_IN_GB_*1024.0f / MEM_PER_DIRECTION_MB;
	   printf("[cuda] : MEM_PER_DIRECTION_MB is %.4f, DIRECTION_CNT_PER_PROCESS %lld using %.4fg memory\n", MEM_PER_DIRECTION_MB, DIRECTION_CNT_PER_PROCESS, DIRECTION_CNT_PER_PROCESS*MEM_PER_DIRECTION_MB/1024);
	   unsigned int * rbuffer = new unsigned int[DIRECTION_CNT_PER_PROCESS*(1u<<(presicion*3-5))];
	   char strBuffer[256];  sprintf(strBuffer, "%s/%d", outputdir.c_str(), (1u << presicion)); outputdir = std::string(strBuffer)+"/";
	   _mkdir(strBuffer);

	   float* dots_dev; size_t num_bytes;
	   struct hipGraphicsResource *cuda_vbo_binder;
	   checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_binder, dotsvbo, cudaGraphicsMapFlagsReadOnly));
	   checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_binder, 0));
	   checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dots_dev, &num_bytes, cuda_vbo_binder));
	   printf("[cuda] : mapped %d bytes dots\n", num_bytes);
	   
	   float* mats_dev;
	   checkCudaErrors(hipMalloc((void**)&mats_dev, sizeof(float)* 16 * rotx*roty));
	   checkCudaErrors(hipMemcpy(mats_dev, mats, sizeof(float)* 16 * rotx*roty, hipMemcpyHostToDevice));

	   unsigned int* result_dev;
	   checkCudaErrors(hipMalloc((void**)&result_dev, DIRECTION_CNT_PER_PROCESS*(1u << (presicion * 3 - 3))));

	   //launch kernels
	   int cid = 0;
	   StopWatchInterface *timer = 0;
	   sdkCreateTimer(&timer);
	   hipDeviceSynchronize();
	   sdkStartTimer(&timer);
	   for (int sid = 0; ;sid += DIRECTION_CNT_PER_PROCESS){
		   int tid = MIN(sid + DIRECTION_CNT_PER_PROCESS - 1, rotx*roty - 1);
		   int blockCnt = ((tid - sid + 1) - 1) / _DIRECTION_CNT_PER_BLOCK_ + 1;

		   transform <<<blockCnt, _THREAD_NUM_PER_BLOCK_ >>>((float3*)dots_dev, dotscnt, dotscenter,mats_dev,sid, tid, (presicion), result_dev);
		   hipDeviceSynchronize();
		   checkCudaErrors(hipMemcpy(rbuffer, result_dev, size_t(tid - sid + 1)*(1u<<(presicion*3-3)),hipMemcpyDeviceToHost));
		   hipDeviceSynchronize();

		   unsigned int *uintptr = rbuffer;
		   unsigned int cnt_per_line = (1u << (presicion - 5));
		   for (int id = sid; id <= tid; id++){
			   int anglex = id % rotx;
			   int angley = id / rotx;
			   sprintf(strBuffer, "x%02d_y%02d.txt", anglex, angley);
			   freopen((outputdir+strBuffer).c_str(),"w",stdout);
			   
			   for (unsigned int ind = 0; ind < (1u << (presicion * 3 - 5)); ind+=cnt_per_line){
				   for (unsigned int step = 0; step < cnt_per_line; step++) printf("%u ",uintptr[ind+step]); puts("");
			   }

			   fclose(stdout);
			   uintptr += (1u << (presicion * 3 - 5));
		   }
		   freopen("CON","w",stdout);

		   printf("[cuda] : iteration %d finish \n", cid++);
		   if (tid >= rotx*roty - 1) break;
	   }
	   sdkStopTimer(&timer);

	   float elapseTime = 1.0e-3 * sdkGetTimerValue(&timer);
	   printf("[cuda] : transform total cost %f s\n",elapseTime);

	   delete rbuffer;
	   sdkDeleteTimer(&timer);
	   checkCudaErrors(hipFree(result_dev));
	   checkCudaErrors(hipFree(mats_dev));
	   checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_binder, 0));
}